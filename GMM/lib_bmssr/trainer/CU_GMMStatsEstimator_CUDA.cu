#include "hip/hip_runtime.h"
// 
// Copyright (c) 2007--2014 Lukas Machlica
// Copyright (c) 2007--2014 Jan Vanek
// 
// University of West Bohemia, Department of Cybernetics, 
// Plzen, Czech Repulic
// 
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
// 1. Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
// 3. The name of the author may not be used to endorse or promote products
//    derived from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
// IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
// IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
// NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
// DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
// THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
// THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE. 

#include "trainer/CU_GMMStatsEstimator_CUDA.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <stdexcept>

// #define GPU_SAFE_MODE // turn on synchronization of kernel executions - slowing down the execution
// #define GPU_CUDA_TIMER // trace execution times of particular kernels
#define GPU_CUDA_CONTEXT_AVAIL

#ifdef GPU_CUDA_CONTEXT_AVAIL
#    include <hip/hip_runtime.h>
#else
#    include <hip/hip_runtime.h>
#endif

#ifdef GPU_CUDA_TIMER
    float GPU_kernel_timer1 = 0;
    float GPU_kernel_timer2 = 0;
    float GPU_kernel_timer3 = 0;
    hipEvent_t start1, start2, start3, stop1, stop2, stop3;
#endif

const unsigned int GMMStatsEstimator_GPU::DIM_BLOCK  = 4;
const unsigned int GMMStatsEstimator_GPU::FRAME_BLOCK = 8;
const unsigned int GMMStatsEstimator_GPU::GAUSS_BLOCK = 32;
const unsigned int GMMStatsEstimator_GPU::DATA_BLOCK = DIM_BLOCK * FRAME_BLOCK;
const unsigned int GMMStatsEstimator_GPU::RESET_BLOCK_SIZE = 128;
const unsigned int GMMStatsEstimator_GPU::MAX_GRID_SIZE = (32*1024-1);
const unsigned int GMMStatsEstimator_GPU::MODELMEM_INCREASE = 100;
const unsigned int GMMStatsEstimator_GPU::MIN_UNUSED_MEM_GPU = 50;
const unsigned int GMMStatsEstimator_GPU::MIN_FRAMES_ON_GPU = 100 * DIM_BLOCK * FRAME_BLOCK;

// include kernels
#include "trainer/CU_GMMStatsEstimator_CUDAkernels.cuh"

template unsigned int GMMStatsEstimator_GPU::alignUP (unsigned int x, unsigned int block2align);
template unsigned int GMMStatsEstimator_GPU::alignDiv (unsigned int x, unsigned int block2align);

GMMStatsEstimator_GPU::GMMStatsEstimator_GPU(int GPU_id) 
: _stats(NULL),
_model(NULL),
_prm(NULL),
_likes(NULL),
_gmodels(NULL),
_gmodelsSize(0),
_gmodelsAllocSize(0),
_sharedMemPerBlock(0),
_pContextGPU(NULL),
_GPU_id(-1)
{
    _opt.minLogLike = -1e+20f;
    _opt.minGamma = 1e-4f;
    _opt.frames_acc_blocks = 8;
    _opt.verbosity = 0;
    _opt.throw_errors = true;

    _model = new model;
    _model->d_means = NULL;
    _model->d_ivars = NULL;
    _model->d_Gconsts = NULL;
    _model->dim = 0;
    _model->dimVar = 0;
    _model->nummix = 0;
    _model->nummix_unaligned = 0;
    _model->nummix_allocS = 0;
    _model->d_modelAlloc = false;
    
    _gmodels = new model* [MODELMEM_INCREASE];
    _gmodelsAllocSize = MODELMEM_INCREASE;
    _gmodels[0] = _model;
    _gmodelsSize = 1;
    
    _prm = new param;
    _prm->d_vecs = NULL;
    _prm->dim = 0;
    _prm->Nframes_unaligned = 0;
    _prm->Nframes = 0;
    _prm->Nframes_allocS = 0;
    _prm->d_paramAlloc = false;

    _stats = new stats;
    _stats->d_meanStats = NULL;
    _stats->statMeanSize = 0;
    _stats->d_varStats = NULL;
    _stats->statProbSize = 0;
    _stats->accFullVar = false;
    _stats->d_mixProb = NULL;
    _stats->statVarSize = 0;
    _stats->d_auxStats = NULL;
    _stats->statAuxSize = 0;        
    _stats->d_statsAlloc = false;

    _likes = new likes;
    _likes->d_gammas = NULL;
    _likes->gammasSize = 0;
    _likes->d_ll = NULL;
    _likes->d_ll_begin = NULL;
    _likes->llSize = 0;
    _likes->ll_storeAll = false;
    _likes->d_totll = NULL;
    _likes->d_aux_ll = NULL;
    
    int devN = deviceCount();    
    if (devN > GPU_id) {
        createContext(GPU_id, &_pContextGPU);

        hipSetDevice(GPU_id);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, GPU_id);
        _sharedMemPerBlock = deviceProp.sharedMemPerBlock;
        _GPU_id = GPU_id;
    }
}



GMMStatsEstimator_GPU::~GMMStatsEstimator_GPU() 
{    
    pushGPU();
    eraseGroups(false);

    eraseModel();
    eraseParam();
    eraseGammas();

    delete _model;
    delete _prm;
    delete _stats;
    delete _likes;

    destroyContext(&_pContextGPU);
}



int GMMStatsEstimator_GPU::deviceCount()
{
    int deviceCount = 0;
    checkError( hipGetDeviceCount(&deviceCount) );
    return deviceCount;
}


unsigned int GMMStatsEstimator_GPU::getSharedMemPerBlock() {
    return _sharedMemPerBlock;
}



void GMMStatsEstimator_GPU::printInfo() {

    printf("\nCUDA GPU Estimator - version 2.0 (1.2.2013)\n");
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if(deviceCount > 0)
        printf("Number of CUDA compatible device(s) found: %d\n", deviceCount);
    else {
        printf("No CUDA compatible device found.\n");
        return;
    }
    
    if(deviceCount < _GPU_id) {
        printf("Device with ID %d was not detected.\n", _GPU_id);
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, _GPU_id);
    printf("Device ID: %d\n", _GPU_id);
    printf("Device name: %s\n", deviceProp.name);
    printf("GPU memory: %d MB\n", deviceProp.totalGlobalMem/1024/1024);
    printf("Core clock: %d MHz\n", deviceProp.clockRate/1000);
    if(strstr(deviceProp.name, "CPU") != NULL) {
        printf("Only GPU emulation available.\n");
    }
    //hipSetDevice(deviceID);
}


void GMMStatsEstimator_GPU::allocOnHost(void **data, unsigned int size) {    
    checkError(  hipHostMalloc(data, size) );
    //*data = (void *) new char [size];    
}



void GMMStatsEstimator_GPU::freeOnHost(void **data) {
    checkError( hipHostFree(*data) );    
    *data = NULL;
    //delete [] *data;    
}




void GMMStatsEstimator_GPU::reallocGroups(unsigned int NGroups) {    

    if (NGroups <= _gmodelsAllocSize) 
        return;

    unsigned int new_size = NGroups + MODELMEM_INCREASE;

    model **tmp_gmodels;
    tmp_gmodels = new model* [new_size];
    memcpy(tmp_gmodels, _gmodels, _gmodelsAllocSize * sizeof(model*));

    delete [] _gmodels;
    _gmodels = tmp_gmodels;

    _gmodelsAllocSize = new_size;
}



void GMMStatsEstimator_GPU::eraseGroups(bool release) 
{
    if (_gmodelsAllocSize < 1) {
        _gmodelsSize = 0;
        return;
    }

    // leave the first model in
    if (_gmodelsSize > 0) 
        _model = _gmodels[0];

    for(unsigned int n = 1; n < _gmodelsSize; n++) 
    {        
        checkError( hipFree(_gmodels[n]->d_means) );
        checkError( hipFree(_gmodels[n]->d_ivars) );
        checkError( hipFree(_gmodels[n]->d_Gconsts) );

        delete _gmodels[n];
        _gmodels[n] = NULL;
    }
    
    if (!release) 
    {
        delete [] _gmodels;
        _gmodels = NULL;
        _gmodelsAllocSize = 0;
    }

    _gmodelsSize = 1;
}



void GMMStatsEstimator_GPU::uploadModel (float* meanBuffC, float* ivarBuffC, float* gConstBuffC, 
                                         unsigned int NMix, unsigned int alignedNMix, unsigned int alignedNMixAlloc, 
                                         unsigned int alignedDim, unsigned int alignedDimVar, bool first)
{
    pushGPU();

    if(first)
        eraseGroups(true);
    else {
        _model = new model;
        _gmodelsSize++;
        reallocGroups(_gmodelsSize);
        _gmodels[_gmodelsSize - 1] = _model;
    }

    if (!_model->d_modelAlloc ||
        alignedNMixAlloc > _model->nummix_allocS || 
        alignedDim != _model->dim || 
        alignedDimVar != _model->dimVar )
    {                    
        allocModel(alignedNMixAlloc, alignedDim, alignedDimVar);
    }    

    checkError( hipMemcpy(_model->d_means, meanBuffC, alignedNMixAlloc * alignedDim * sizeof(float), hipMemcpyHostToDevice) );
    checkError( hipMemcpy(_model->d_ivars, ivarBuffC, alignedNMixAlloc * alignedDimVar * sizeof(float), hipMemcpyHostToDevice) );
    checkError( hipMemcpy(_model->d_Gconsts, gConstBuffC, alignedNMixAlloc * sizeof(float), hipMemcpyHostToDevice) );
    
    _model->dim = alignedDim;
    _model->dimVar = alignedDimVar;
    _model->nummix = alignedNMix;
    _model->nummix_unaligned = NMix;
    _model->nummix_allocS = alignedNMixAlloc;    

    popGPU();
}



void GMMStatsEstimator_GPU::eraseModel() {    

    checkError( hipFree(_model->d_means) );
    _model->d_means = NULL;

    checkError( hipFree(_model->d_ivars) );
    _model->d_ivars = NULL;

    checkError( hipFree(_model->d_Gconsts) );
    _model->d_Gconsts = NULL;

    _model->nummix_allocS = 0;
    _model->nummix = 0;
    _model->dim = 0;

    _model->d_modelAlloc = false;
#ifdef GPU_CUDA_TIMER
    //init events for GPU timers
    hipEventDestroy(start1);
    hipEventDestroy(start2);
    hipEventDestroy(start3);
    hipEventDestroy(stop1);
    hipEventDestroy(stop2);
    hipEventDestroy(stop3);

    //print total elapsed times
    printf("GPU elapsed time in kernel 1 : %6.3f\n", GPU_kernel_timer1);
    printf("GPU elapsed time in kernel 2 : %6.3f\n", GPU_kernel_timer2);
    printf("GPU elapsed time in kernel 3 : %6.3f\n", GPU_kernel_timer3);
    printf("Total GPU time in all kernels: %6.3f\n", GPU_kernel_timer1+GPU_kernel_timer2+GPU_kernel_timer3);
#endif    
}



void GMMStatsEstimator_GPU::setModelToBeUsed(unsigned int n) 
{
#ifndef _EXCLUDE_SAFETY_CONDS_
    if (n > _gmodelsSize)
        throw std::runtime_error("setModelToBeUsed(): requested model out of bounds!");
#endif

    pushGPU();

    _model = _gmodels[n];

    popGPU();
}



void GMMStatsEstimator_GPU::allocModel(unsigned int alignedNMix, unsigned int alignedDim, unsigned int alignedDimVar) 
{
    eraseModel();

    checkError( hipMalloc((void **)& _model->d_means, alignedNMix * alignedDim * sizeof(float)) );
    checkError( hipMalloc((void **)& _model->d_ivars, alignedNMix * alignedDimVar * sizeof(float)) );
    checkError( hipMalloc((void **)& _model->d_Gconsts, alignedNMix * sizeof(float)) );    

    _model->nummix_allocS = alignedNMix;
    _model->dim = alignedDim;
    _model->dimVar = alignedDimVar;

    _model->d_modelAlloc = true;
#ifdef GPU_CUDA_TIMER
    //init events for GPU timers
    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&start3);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);
    hipEventCreate(&stop3);
    GPU_kernel_timer1 = 0.0f;
    GPU_kernel_timer2 = 0.0f;
    GPU_kernel_timer3 = 0.0f;
#endif
}



void GMMStatsEstimator_GPU::eraseParam() 
{
    checkError( hipFree(_prm->d_vecs) );
    
    _prm->d_vecs = NULL;

    _prm->Nframes_allocS = 0;
    _prm->Nframes = 0;
    _prm->dim = 0;

    _prm->d_paramAlloc = false;
}



void GMMStatsEstimator_GPU::allocParam(unsigned int alignedDim, unsigned int NframesAligned) 
{
    eraseParam();

    checkError( hipMalloc((void **)& _prm->d_vecs, NframesAligned * alignedDim * sizeof(float)) );

    _prm->Nframes_allocS = NframesAligned;
    _prm->dim = alignedDim;

    _prm->d_paramAlloc = true;
}



void GMMStatsEstimator_GPU::uploadParam (float* alignedVecs, unsigned int alignedDim, 
                                         unsigned int Nframes, unsigned int Nframes2processAtOnce,
                                         bool requestLogLikeForEachFrame) 
{
    pushGPU();

    unsigned int NframesAligned = alignUP<unsigned int> (Nframes, FRAME_BLOCK);
    unsigned int NframesAlignedPOnce = alignUP<unsigned int> (Nframes2processAtOnce, FRAME_BLOCK);    

#ifndef _EXCLUDE_SAFETY_CONDS_
    if (!_model->d_modelAlloc)
        throw std::runtime_error("uploadParam(): insert model first!");
    if (_model->dim != alignedDim)
        throw std::runtime_error("uploadParam(): dimension mismatch!");
#endif

    // if allocated but too small => realloc
    if (!_prm->d_paramAlloc || 
        NframesAligned > _prm->Nframes_allocS || 
        alignedDim != _prm->dim)
    {
        allocParam(alignedDim, NframesAligned);
    }
            
    // if likelihood for each frame should be kept
    unsigned int ext_ll_size = (unsigned int) requestLogLikeForEachFrame * (NframesAligned - NframesAlignedPOnce);
    if ((NframesAlignedPOnce + ext_ll_size) > _likes->llSize || 
        NframesAlignedPOnce * _model->nummix_allocS > _likes->gammasSize)
    {        
        allocGammas(NframesAlignedPOnce, _model->nummix_allocS, ext_ll_size);
        _likes->ll_storeAll = requestLogLikeForEachFrame;
    }

    checkError( hipMemcpy(_prm->d_vecs, alignedVecs, NframesAligned * alignedDim * sizeof(float), hipMemcpyHostToDevice) );
    
    _prm->Nframes_processed = NframesAlignedPOnce;
    _prm->Nframes_unaligned = Nframes;
    _prm->Nframes = NframesAligned;
    popGPU();
}



unsigned int GMMStatsEstimator_GPU::getNSamplesOnGPU() {
    return _prm->Nframes_unaligned;
}



void GMMStatsEstimator_GPU::eraseGammas() {

    checkError( hipFree(_likes->d_gammas) );

    _likes->d_gammas = NULL;
    _likes->gammasSize = 0;

    checkError( hipFree(_likes->d_ll) );
    checkError( hipFree(_likes->d_totll) );
    checkError( hipFree(_likes->d_aux_ll) );
    
    _likes->d_totll = NULL;    
    _likes->d_ll = NULL;
    _likes->d_ll_begin = NULL;
    
    _likes->d_aux_ll = NULL;

    _likes->llSize = 0;
    _likes->ll_storeAll = false;
}



void GMMStatsEstimator_GPU::allocGammas(unsigned int alignedNframes, unsigned int nummix_allocS, unsigned int ext_ll_size) {
    
    eraseGammas();

    unsigned int alignedSize = alignedNframes * nummix_allocS;

    checkError( hipMalloc((void **)& _likes->d_gammas, alignedSize * sizeof(float)) );
    checkError( hipMalloc((void **)& _likes->d_ll, (alignedNframes + ext_ll_size) * sizeof(float)) );
    checkError( hipMalloc((void **)& _likes->d_totll, sizeof(float)) );
    checkError( hipMalloc((void **)& _likes->d_aux_ll, (alignedNframes) * sizeof(float)) );

    _likes->d_ll_begin = _likes->d_ll;
    _likes->gammasSize = alignedSize;
    _likes->llSize = alignedNframes + ext_ll_size;
}



void GMMStatsEstimator_GPU::allocAccStats (unsigned int alignedNMix, unsigned int alignedDim, 
                                           bool fullVarAcc, bool allocAuxStats) 
{
    eraseAccStats();
    
    if (allocAuxStats)
        checkError( hipMalloc((void **)& _stats->d_auxStats, _opt.frames_acc_blocks * alignedNMix * (alignedDim + 2) * sizeof(float)) );

    checkError( hipMalloc((void **)& _stats->d_meanStats, _opt.frames_acc_blocks * alignedNMix * alignedDim * sizeof(float)) );

    unsigned int varDim = alignedDim;
    if(fullVarAcc) {
        varDim = (alignedDim * (alignedDim + DIM_BLOCK)) / 2;    // the diagonal part has to be alligned according to DIM_BLOCK
        _stats->accFullVar = true;
    }

    checkError( hipMalloc((void **)& _stats->d_varStats, _opt.frames_acc_blocks * alignedNMix * varDim * sizeof(float)) );
    checkError( hipMalloc((void **)& _stats->d_mixProb, _opt.frames_acc_blocks * alignedNMix * sizeof(float)) );    

    _stats->statMeanSize = alignedNMix * alignedDim;
    _stats->statVarSize = alignedNMix * varDim;    
    _stats->statProbSize = alignedNMix;
    if (allocAuxStats)
        _stats->statAuxSize = alignedNMix * (alignedDim + 2);

    _stats->d_statsAlloc = true;
}



void GMMStatsEstimator_GPU::eraseAccStats() {

    checkError( hipFree(_stats->d_meanStats) );
    checkError( hipFree(_stats->d_varStats) );
    checkError( hipFree(_stats->d_mixProb) );
    checkError( hipFree(_stats->d_auxStats) );

    _stats->d_meanStats = NULL;
    _stats->d_varStats = NULL;
    _stats->d_mixProb = NULL;
    _stats->d_auxStats = NULL;

    _stats->statAuxSize = 0;
    _stats->statMeanSize = 0;
    _stats->statVarSize = 0;
    _stats->statProbSize = 0;
    _stats->accFullVar = false;

    _stats->d_statsAlloc = false;
}


void GMMStatsEstimator_GPU::resetAccStats() {

    if (_stats->statAuxSize > 0) {
        hipMemsetAsync(_stats->d_auxStats, 0, sizeof(float) * _stats->statAuxSize * _opt.frames_acc_blocks);
        //setZeroArray <<< _stats->statAuxSize/RESET_BLOCK_SIZE, RESET_BLOCK_SIZE >>>
        //    (_stats->d_auxStats, _stats->statAuxSize);
    }
    if (_stats->statMeanSize > 0) {
        hipMemsetAsync(_stats->d_meanStats, 0, sizeof(float) * _stats->statMeanSize * _opt.frames_acc_blocks);
        //setZeroArray <<< _stats->statMeanSize/RESET_BLOCK_SIZE, RESET_BLOCK_SIZE >>>
        //    (_stats->d_meanStats, _stats->statMeanSize);
    }
    if (_stats->statVarSize > 0) {
        hipMemsetAsync(_stats->d_varStats, 0, sizeof(float) * _stats->statVarSize * _opt.frames_acc_blocks);
        //setZeroArray <<< _stats->statVarSize/RESET_BLOCK_SIZE, RESET_BLOCK_SIZE >>>
        //    (_stats->d_varStats, _stats->statVarSize);
    }
    if (_stats->statProbSize > 0) {
        hipMemsetAsync(_stats->d_mixProb, 0, sizeof(float) * _stats->statProbSize * _opt.frames_acc_blocks);
        //setZeroArray <<< _stats->statProbSize/RESET_BLOCK_SIZE, RESET_BLOCK_SIZE >>>
        //    (_stats->d_mixProb, _stats->statProbSize);
    }
    if (_likes->d_totll != NULL)
        checkError( hipMemsetAsync(_likes->d_totll, 0, sizeof(float)) );

#ifdef GPU_SAFE_MODE    
    hipDeviceSynchronize();
    checkError( hipGetLastError() );
#endif
}



void GMMStatsEstimator_GPU::compLogLikes() 
{
    pushGPU();
    
#ifndef _EXCLUDE_SAFETY_CONDS_
    if (!_model->d_modelAlloc || !_prm->d_paramAlloc)
        throw std::runtime_error("getLogLikes(): none data!");
    if (_prm->Nframes > _likes->llSize || _model->dim != _prm->dim)
        throw std::runtime_error("getLogLikes(): dimension or #sample mismatch!");
#endif
    
    unsigned int NframeBlocks = alignDiv<unsigned int> (_prm->Nframes_processed / FRAME_BLOCK, MAX_GRID_SIZE);
    unsigned int shiftFrames = 0;
    for (unsigned int iBlock = 0; iBlock < NframeBlocks; iBlock++) 
    {            
        unsigned int NSamplesTmp = ((iBlock+1) * _prm->Nframes_processed > _prm->Nframes) ? _prm->Nframes - iBlock * _prm->Nframes_processed : _prm->Nframes_processed;

        compLogLikeAndGammas(shiftFrames, NSamplesTmp);            
        shiftFrames += NSamplesTmp;
    }
    
    popGPU();
}



void GMMStatsEstimator_GPU::getLogLikes (float* outLogLikes) 
{
    pushGPU();    

#ifndef _EXCLUDE_SAFETY_CONDS_
    if (_prm->Nframes_processed < _prm->Nframes && !_likes->ll_storeAll)
        throw std::runtime_error("getLogLikes(): not all the loglikes were stored -> set 'll_storeAll' = true before uploading param!");
#endif
    
    checkError( hipMemcpy(outLogLikes, _likes->d_ll_begin, _prm->Nframes_unaligned *  sizeof(float), hipMemcpyDeviceToHost) );
    popGPU();
}



void GMMStatsEstimator_GPU::compLogLikeAndGammas(unsigned int shiftFrames, unsigned int alignedNframes) 
{
    
#ifdef GPU_CUDA_TIMER
    hipEventRecord( start1, 0 );
#endif

    // estimate raw (unnormalized) gammas (store in _likes->d_gammas) for each frame in _prm->d_vecs    
    if(_model->dimVar != _model->dim) {        
        gammasKernelFull <<< dim3(alignedNframes/FRAME_BLOCK, _model->nummix/GAUSS_BLOCK), GAUSS_BLOCK, FRAME_BLOCK*_model->dim*sizeof(float) >>> 
            (*_model, *_prm, *_likes, shiftFrames);    
    }
    else {        
        if(_prm->dim <= 128) {
            gammasKernel <<< dim3(alignedNframes/FRAME_BLOCK, _model->nummix/GAUSS_BLOCK), GAUSS_BLOCK, FRAME_BLOCK*_model->dim*sizeof(float) >>> (*_model, *_prm, *_likes, shiftFrames);
        } else {
            const unsigned int DIMS_PER_BLOCK = 1024/DATA_BLOCK;
            gammasKernelLargeDim <<< dim3(alignedNframes/FRAME_BLOCK, _model->nummix/GAUSS_BLOCK), GAUSS_BLOCK, FRAME_BLOCK*DIMS_PER_BLOCK*DIM_BLOCK*sizeof(float) >>> (*_model, *_prm, *_likes, shiftFrames);
        }
    }

#ifdef GPU_CUDA_TIMER
    hipEventRecord( stop1, 0 );
#endif

#ifdef GPU_SAFE_MODE
    hipDeviceSynchronize();
    checkError( hipGetLastError() );
#endif

#ifdef GPU_CUDA_TIMER
    hipEventRecord( start2, 0 );
#endif

    // sum raw (unnormalized) gammas for each frame along all the gaussians;
    // note: data stored in float4 => #blocks = alignedLlsSize/4    
    _likes->d_ll = _likes->d_ll_begin + _likes->ll_storeAll * (shiftFrames / 4);
    logLikeKernel <<< alignedNframes/4, GAUSS_BLOCK >>> 
        (*_likes, _model->nummix_unaligned, _model->nummix, _opt.minLogLike);
    
#ifdef GPU_CUDA_TIMER
    hipEventRecord( stop2, 0 );
#endif

#ifdef GPU_SAFE_MODE
    hipDeviceSynchronize();
    checkError( hipGetLastError() );
#endif
}



// ! assuming that gammas & logLikes were already computed using compLogLikeAndGammas()
// note: size of gammas   = NSamples * number_of_mixtures
//         size of loglikes = NSamples
void GMMStatsEstimator_GPU::getGammasAndLL (float *gammas, float *loglikes,
                                            unsigned int alignedNframes, unsigned int alignedNmix)
{
    pushGPU();

#ifndef _EXCLUDE_SAFETY_CONDS_    
    if (alignedNframes > _prm->Nframes_processed || alignedNmix > _model->nummix)
        throw std::runtime_error("getGammasAndLL(): nummix or #sample mismatch!!");
    if (_prm->Nframes_processed < _prm->Nframes)
        throw std::runtime_error("getGammasAndLL(): not all the gammas and loglikes were stored -> mask unavailable!");
#endif
    
    checkError( hipMemcpy(gammas, _likes->d_gammas, alignedNmix * alignedNframes * sizeof(float), hipMemcpyDeviceToHost) );
    checkError( hipMemcpy(loglikes, _likes->d_ll_begin, alignedNframes * sizeof(float), hipMemcpyDeviceToHost) );
    popGPU();
}



void GMMStatsEstimator_GPU::compAccStats (bool fullVarAcc, bool auxStats) 
{
#ifndef _EXCLUDE_SAFETY_CONDS_    
    if (!_model->d_modelAlloc || !_prm->d_paramAlloc)
        throw std::runtime_error("compAccStats(): none data");
    if (_model->dim != _prm->dim)
        throw std::runtime_error("compAccStats(): dimension mismatch");
#endif

    pushGPU();

    // cannot be moved to uploadModel() since auxStats and fullVarAcc has not to be known in advance
    if ( _model->nummix_allocS * _model->dim > _stats->statMeanSize || 
         (fullVarAcc && !_stats->accFullVar) )
    {
        allocAccStats (_model->nummix_allocS, _model->dim, fullVarAcc, auxStats);
    }
    resetAccStats();
        
    unsigned int NframeBlocks = alignDiv<unsigned int> (_prm->Nframes_unaligned, _prm->Nframes_processed);
    unsigned int shiftFrames = 0;
    for (unsigned int iBlock = 0; iBlock < NframeBlocks; iBlock++) 
    {            
        unsigned int NSamplesTmp = ((iBlock+1) * _prm->Nframes_processed > _prm->Nframes) ? _prm->Nframes - iBlock * _prm->Nframes_processed : _prm->Nframes_processed;
        
        accumulateStats(shiftFrames, NSamplesTmp, auxStats, fullVarAcc);
        shiftFrames += NSamplesTmp;        
    }
    
    if(_opt.frames_acc_blocks > 0)
        sumAccStats(auxStats);

    popGPU();
}



//void GMMStatsEstimator_GPU::sumAccStats(bool auxStats)
//{
//    int numBlocks;
//    numBlocks = (_stats->statMeanSize / _model->nummix_allocS * _model->nummix) / DIM_BLOCK / GAUSS_BLOCK;
//    addArrays4Kernel <<< numBlocks, GAUSS_BLOCK >>>
//        (_stats->d_meanStats, _opt.frames_acc_blocks, _stats->statMeanSize / _model->nummix_allocS * _model->nummix);
//
//#ifdef GPU_SAFE_MODE
//        hipDeviceSynchronize();
//        checkError( hipGetLastError() );
//#endif
//    
//    numBlocks = (_stats->statVarSize / _model->nummix_allocS * _model->nummix) / DIM_BLOCK / GAUSS_BLOCK;
//    addArrays4Kernel <<< numBlocks, GAUSS_BLOCK >>>
//        (_stats->d_varStats, _opt.frames_acc_blocks, _stats->statVarSize / _model->nummix_allocS * _model->nummix);
//
//#ifdef GPU_SAFE_MODE
//        hipDeviceSynchronize();
//        checkError( hipGetLastError() );
//#endif
//
//    numBlocks = (_stats->statProbSize / _model->nummix_allocS * _model->nummix) / GAUSS_BLOCK;
//    addArraysKernel <<< numBlocks, GAUSS_BLOCK >>>
//        (_stats->d_mixProb, _opt.frames_acc_blocks, _stats->statProbSize / _model->nummix_allocS * _model->nummix);
//
//#ifdef GPU_SAFE_MODE
//        hipDeviceSynchronize();
//        checkError( hipGetLastError() );
//#endif
//
//    //LLLLLLLLLL
//    printfCUDADATAfloat(_stats->d_mixProb, 256, "\n");
//
//    if (auxStats)
//    {
//        numBlocks = (_stats->statAuxSize / _model->nummix_allocS * _model->nummix) / GAUSS_BLOCK;
//        addArraysKernel <<< numBlocks, GAUSS_BLOCK >>>
//            (_stats->d_auxStats, _opt.frames_acc_blocks, _stats->statAuxSize / _model->nummix_allocS * _model->nummix);
//    }
//
//#ifdef GPU_SAFE_MODE
//        hipDeviceSynchronize();
//        checkError( hipGetLastError() );
//#endif
//}

void GMMStatsEstimator_GPU::sumAccStats(bool auxStats)
{
    addArrays4Kernel <<< _stats->statMeanSize / DIM_BLOCK / GAUSS_BLOCK, GAUSS_BLOCK >>>
        (_stats->d_meanStats, _opt.frames_acc_blocks, _stats->statMeanSize);

#ifdef GPU_SAFE_MODE
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif


    addArrays4Kernel <<< _stats->statVarSize / DIM_BLOCK / GAUSS_BLOCK, GAUSS_BLOCK >>>
        (_stats->d_varStats, _opt.frames_acc_blocks, _stats->statVarSize);

#ifdef GPU_SAFE_MODE
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

    addArraysKernel <<< _stats->statProbSize / GAUSS_BLOCK, GAUSS_BLOCK >>>
        (_stats->d_mixProb, _opt.frames_acc_blocks, _stats->statProbSize);

#ifdef GPU_SAFE_MODE
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

    if (auxStats)
    {
        addArraysKernel <<< _stats->statAuxSize / GAUSS_BLOCK, GAUSS_BLOCK >>>
            (_stats->d_auxStats, _opt.frames_acc_blocks, _stats->statAuxSize);

#ifdef GPU_SAFE_MODE
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif
    }
}

void GMMStatsEstimator_GPU::getAccStats (float &totLL, float *meanStats, float *varStats, float *mixProbs, float* auxStats, 
                                         bool fullVarAcc) 
{
    pushGPU();    
    
    checkError( hipMemcpy(&totLL, _likes->d_totll, sizeof(float), hipMemcpyDeviceToHost) );

    if(meanStats != NULL)
        checkError( hipMemcpy(meanStats, _stats->d_meanStats, _model->nummix_unaligned * _prm->dim * sizeof(float), hipMemcpyDeviceToHost) );
    
    if (varStats != NULL) {
        unsigned int aVarDim = fullVarAcc ? (_prm->dim * (_prm->dim + DIM_BLOCK)) / 2 : _prm->dim;
        checkError( hipMemcpy(varStats, _stats->d_varStats, _model->nummix_unaligned * aVarDim * sizeof(float), hipMemcpyDeviceToHost) );
    }

    if(mixProbs != NULL)
        checkError( hipMemcpy(mixProbs, _stats->d_mixProb, _model->nummix_unaligned * sizeof(float), hipMemcpyDeviceToHost) );

    if(auxStats != NULL)
        checkError( hipMemcpy(auxStats, _stats->d_auxStats, _model->nummix_unaligned * (_prm->dim + 2) * sizeof(float), hipMemcpyDeviceToHost) );
    
    popGPU();
}



void GMMStatsEstimator_GPU::accumulateStats(unsigned int shiftFrames, unsigned int NSamplesTmp, 
                                            bool accAuxStats, bool fullVarAcc)
{
    compLogLikeAndGammas(shiftFrames, NSamplesTmp);
    
    //LLLLLLLLLLLLLL
    //printfCUDADATAfloat((float*) _likes->d_gammas, 135); 

    gammasNormKernel <<< NSamplesTmp/FRAME_BLOCK, GAUSS_BLOCK >>> 
        (*_likes, _opt, _model->nummix, _prm->Nframes_unaligned - shiftFrames);

#ifdef GPU_SAFE_MODE        
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

    //LLLLLLLLLLLLLL
//        {
//        FILE *fid = fopen("xx.txt", "w");
//        float *foo = new float[NSamplesTmp*_model->nummix];
//        GMMStatsEstimator_GPU::checkError( hipMemcpy(foo, _likes->d_gammas, sizeof(float)*NSamplesTmp*_model->nummix, hipMemcpyDeviceToHost) );
//    for(int i = 0; i<NSamplesTmp*_model->nummix; i++) {
//        fprintf(fid, "%f\n", foo[i]);
//    }
//    delete foo;
//    fclose(fid);
//}

#ifdef GPU_CUDA_TIMER
    hipEventRecord( start3, 0 );
#endif

    if(!fullVarAcc)
    {
        accStatsDiagKernel <<< dim3(_model->dim/DIM_BLOCK, _model->nummix/GAUSS_BLOCK * _opt.frames_acc_blocks), GAUSS_BLOCK >>> 
            (*_prm, *_likes, *_stats, _model->nummix, shiftFrames);
    }
    else 
    {
        unsigned int dimVarDBxDB = _model->dimVar / (DIM_BLOCK * DIM_BLOCK);
        accStatsFullKernel <<< dim3(_model->nummix_unaligned, _opt.frames_acc_blocks), dimVarDBxDB >>> 
            (*_prm, *_likes, *_stats, _opt, _model->nummix, shiftFrames);
    }

#ifdef GPU_SAFE_MODE        
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

        //LLLLLLLLLLLLLL
        //printfCUDADATAfloat((float*) _stats->d_mixProb, 256); 


    if(accAuxStats) {

        normAuxGammasKernel <<< NSamplesTmp/4, GAUSS_BLOCK >>> (*_likes, _model->nummix_unaligned, _model->nummix, _opt.minGamma);

#ifdef GPU_SAFE_MODE        
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

        //LLLLLLLLLLLLLL
        //printfCUDADATAfloat((float*) _likes->d_aux_ll, 8); 
        //printfCUDADATAfloat((float*) _likes->d_gammas, 8); 
        //dumpCUDADATAfloat((float*) _likes->d_aux_ll, 837, 1, "gammas_norm_cuda.txt"); 

        accAuxStatsKernel <<< dim3(_model->dim/DIM_BLOCK, _model->nummix/GAUSS_BLOCK * _opt.frames_acc_blocks), dim3(GAUSS_BLOCK / DIM_BLOCK, DIM_BLOCK) >>> 
            (*_prm, *_likes, *_stats, _opt, _model->nummix, shiftFrames);

#ifdef GPU_SAFE_MODE        
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif

        //LLLLLLLLLLLLLL
        //printfCUDADATAfloat((float*) _stats->d_auxStats, 38*2);
        //dumpCUDADATAfloat((float*) _stats->d_auxStats, 32*38, 8, "aux_cuda.txt");

    }

#ifdef GPU_CUDA_TIMER
    hipEventRecord( stop3, 0 );
#endif

    unsigned int NS_GB_aligned = alignUP<unsigned int> (NSamplesTmp, 4*GAUSS_BLOCK);     
    sumLogLike <<< NS_GB_aligned / GAUSS_BLOCK, GAUSS_BLOCK >>>
        (*_likes, _prm->Nframes_unaligned - shiftFrames, NSamplesTmp);

#ifdef GPU_SAFE_MODE        
        hipDeviceSynchronize();
        checkError( hipGetLastError() );
#endif    

#ifdef GPU_CUDA_TIMER
    //update GPU timers
    float time;
    hipEventElapsedTime( &time, start1, stop1 );
    GPU_kernel_timer1 += time;
    hipEventElapsedTime( &time, start2, stop2 );
    GPU_kernel_timer2 += time;
    hipEventElapsedTime( &time, start3, stop3 );
    GPU_kernel_timer3 += time;
#endif
}



void GMMStatsEstimator_GPU::createContext(int GPU_id, void **pctx) 
{
#ifdef GPU_CUDA_CONTEXT_AVAIL
    hipDevice_t device;
    *pctx = (void *) new hipCtx_t();
    
    checkError( hipInit(0) );
    checkError( hipDeviceGet(&device, GPU_id) );
    checkError( hipCtxCreate((hipCtx_t *) (*pctx), hipDeviceScheduleAuto, device) );
#endif
}


void GMMStatsEstimator_GPU::destroyContext(void **pctx) 
{       
#ifdef GPU_CUDA_CONTEXT_AVAIL
    checkError( hipCtxDestroy(*(hipCtx_t *) (*pctx)) );
        
    delete (hipCtx_t *) (*pctx);    
    *pctx = NULL;
#endif
}


void GMMStatsEstimator_GPU::popGPU() 
{
#ifdef GPU_CUDA_CONTEXT_AVAIL
    checkError( hipCtxPopCurrent(NULL) );
#endif
}


void GMMStatsEstimator_GPU::pushGPU()
{      
#ifdef GPU_CUDA_CONTEXT_AVAIL
    checkError( hipCtxPushCurrent(* (hipCtx_t *) _pContextGPU) );
#else
    checkError( hipSetDevice(_GPU_id) );
#endif
}



template <class T>
T GMMStatsEstimator_GPU::alignUP(T x, T block2align) 
{
    return block2align * (x / block2align + (x % block2align > 0));        
}



template <class T>
T GMMStatsEstimator_GPU::alignDiv(T x, T block2align) 
{
    return (x / block2align + (x % block2align > 0));
}



bool GMMStatsEstimator_GPU::checkError(int error, int verbosity, bool throw_errors)
{    
    hipError_t err = (hipError_t) error;
    if(err != hipSuccess) 
    {
        std::stringstream e;
        e <<  "CUDA Error #" << e << ": " << hipGetErrorString(err) << " (file: " << __FILE__ << ")";
        
        if (throw_errors)
            throw std::runtime_error(e.str().c_str());

        if(verbosity > 0)
            std::cout << e.str() << std::endl;

        return false;
    }
    else return true;
}



//void GMMStatsEstimator_GPU::getNSamples2GPU (unsigned int &NSamplesGPU, unsigned int &NSamplesGPUprocess, 
//                                             unsigned int NSamples, unsigned int dim, 
//                                             unsigned int NAccBlocks, unsigned int aligDimAux,
//                                             bool allocStats, float maxGPUMem2UseGB)
//{        
//#ifndef _EXCLUDE_SAFETY_CONDS_    
//    if (!_model->d_modelAlloc)
//        throw std::runtime_error("getNSamples2GPU(): insert model first!");
//#endif
//
//    pushGPU();    
//    
//    size_t maxGPUMem2Use = static_cast<size_t> (maxGPUMem2UseGB * 1024.0f * 1024.0f * 1024.0f);    
//
//    size_t freeMem = 0, totMem = 0;
//#ifdef GPU_CUDA_CONTEXT_AVAIL
//    checkError( hipMemGetInfo(&freeMem, &totMem) );
//    if (freeMem < maxGPUMem2Use)
//        maxGPUMem2Use = freeMem;
//#else
//    hipDeviceProp_t deviceProp;
//    checkError( hipGetDeviceProperties(&deviceProp, _GPU_id) );
//    totMem = deviceProp.totalGlobalMem;
//    if (maxGPUMem2Use == 0)
//        maxGPUMem2Use = 0.7 * totMem;
//#endif
//
//    if (maxGPUMem2Use > totMem)
//        maxGPUMem2Use = 0.7 * totMem;
//
//    if (maxGPUMem2Use > 0)
//        freeMem = maxGPUMem2Use;
//
//    size_t min_unused_memory = MIN_UNUSED_MEM_GPU << 20;
//    size_t min_frames_buff_size = DIM_BLOCK * FRAME_BLOCK << 20;
//
//    size_t min_Nframes = min_frames_buff_size / dim / sizeof(float);
//    //size_t mem_SampCPU = NSamples * dim * sizeof(float);
//
//    size_t mem_GammasGPU = (_model->nummix_allocS + 1) * min_Nframes * sizeof(float);
//    size_t mem_StatsGPU = 0;
//    if (allocStats)
//        mem_StatsGPU = NAccBlocks * ( aligDimAux + _model->nummix_allocS * (1 + _model->dim + _model->dimVar) ) * sizeof(float);
//
//    size_t toAllocGPU = mem_StatsGPU + mem_GammasGPU + min_frames_buff_size;
//
//    if (toAllocGPU + min_unused_memory > freeMem)
//        throw std::runtime_error("getNSamples2GPU(): not enough free memory on GPU");
//
//    size_t memGPUAvailable = freeMem - mem_StatsGPU - min_unused_memory;
//
//    // how many frames can be allocated on GPU along with gammas
//    size_t mem_per_frame = (size_t) ((_model->nummix_allocS + 1) * sizeof(float) + _model->dim * sizeof(float));
//    
//    // NSamplesGPU = avail_mem / mem_for_1_frame_and_adjacent_gammas
//    NSamplesGPU = memGPUAvailable / mem_per_frame;
//
//    if (NSamplesGPU > NSamples)
//        NSamplesGPU = NSamples;
//
//    // align to FRAME_BLOCK
//    size_t NGPUa = alignUP<size_t> (NSamplesGPU, FRAME_BLOCK);        
//
//    // test whether it is possible to process all the frames (max grid size to small?)
//    NSamplesGPUprocess = NSamplesGPU;
//    if (NGPUa / FRAME_BLOCK > MAX_GRID_SIZE) 
//    {
//        // how many frames fit into the grid
//        size_t max_memSGridGPU = MAX_GRID_SIZE * FRAME_BLOCK * mem_per_frame;
//        memGPUAvailable -= max_memSGridGPU;
//
//        size_t NS_res = memGPUAvailable / (_model->dim * sizeof(float));
//
//        NSamplesGPU = MAX_GRID_SIZE * FRAME_BLOCK + NS_res;
//        NSamplesGPUprocess = MAX_GRID_SIZE * FRAME_BLOCK;
//    }    
//
//    popGPU();
//}
void GMMStatsEstimator_GPU::getNSamples2GPU (unsigned int &NSamplesGPU, unsigned int &NSamplesGPUprocess, 
                                             unsigned int NSamples, unsigned int dim, 
                                             unsigned int NAccBlocks, unsigned int aligDimAux,
                                             bool allocStats, float maxGPUMem2UseGB)
{        
#ifndef _EXCLUDE_SAFETY_CONDS_    
    if (!_model->d_modelAlloc)
        throw std::runtime_error("getNSamples2GPU(): insert model first!");
#endif

    pushGPU();    
    
    size_t maxGPUMem2Use = static_cast<size_t> (maxGPUMem2UseGB * 1024.0f * 1024.0f);    // KB

    size_t freeMem = 0, totMem = 0;
#ifdef GPU_CUDA_CONTEXT_AVAIL
    checkError( hipMemGetInfo(&freeMem, &totMem) );
    freeMem >>= 10; // KB
    totMem >>= 10; // KB
    if (freeMem < maxGPUMem2Use)
        maxGPUMem2Use = freeMem;
#else
    hipDeviceProp_t deviceProp;
    checkError( hipGetDeviceProperties(&deviceProp, _GPU_id) );
    totMem = deviceProp.totalGlobalMem;
    totMem >>= 10; // KB
    if (maxGPUMem2Use == 0)
        maxGPUMem2Use = 0.7 * totMem;
#endif

    if (maxGPUMem2Use > totMem)
        maxGPUMem2Use = 0.7 * totMem;

    if (maxGPUMem2Use > 0)
        freeMem = maxGPUMem2Use;

    size_t min_unused_memory = MIN_UNUSED_MEM_GPU << 10; // KB

    size_t min_Nframes = MIN_FRAMES_ON_GPU;
    size_t min_frames_buff_size = min_Nframes * dim * sizeof(float);
    min_frames_buff_size >>= 10; // KB

    size_t mem_GammasGPU = (_model->nummix_allocS + 2) * min_Nframes * sizeof(float);
    mem_GammasGPU >>= 10; // KB
    size_t mem_StatsGPU = 0;
    if (allocStats) {
        mem_StatsGPU = NAccBlocks * ( aligDimAux + _model->nummix_allocS * (1 + _model->dim + _model->dimVar) ) * sizeof(float);
        mem_StatsGPU >>= 10; // KB        
    }

    size_t toAllocGPU = mem_StatsGPU + mem_GammasGPU + min_frames_buff_size; // KB    

    if (toAllocGPU + min_unused_memory > freeMem)
        throw std::runtime_error("getNSamples2GPU(): not enough free memory on GPU");

    size_t memGPUAvailable = freeMem - mem_StatsGPU - min_unused_memory; // KB

    // how many frames can be allocated on GPU along with gammas
    size_t mem_per_frame = (size_t) ( (_model->nummix_allocS + 1 + _model->dim) * sizeof(float) ); // B
    
    // NSamplesGPU = avail_mem / mem_for_1_frame_and_adjacent_gammas
    NSamplesGPU = 1024 * (memGPUAvailable / mem_per_frame);

    if (NSamplesGPU > NSamples)
        NSamplesGPU = NSamples;

    // align to FRAME_BLOCK
    size_t NGPUa = alignUP<size_t> (NSamplesGPU, FRAME_BLOCK);        

    // test whether it is possible to process all the frames (max grid size to small?)
    NSamplesGPUprocess = NSamplesGPU;
    if (NGPUa / FRAME_BLOCK > MAX_GRID_SIZE) 
    {
        // how many frames fit into the grid        
        size_t max_memSGridGPU = MAX_GRID_SIZE * mem_per_frame;
        max_memSGridGPU >>= 10; // KB
        max_memSGridGPU *= FRAME_BLOCK;
        memGPUAvailable -= max_memSGridGPU;

        size_t NS_res = 1024 * (memGPUAvailable / (_model->dim * sizeof(float)));

        NSamplesGPU = MAX_GRID_SIZE * FRAME_BLOCK + NS_res;
        NSamplesGPUprocess = MAX_GRID_SIZE * FRAME_BLOCK;
    }
    
    if (NSamplesGPU > NSamples)
        NSamplesGPU = NSamples;

    popGPU();
}
